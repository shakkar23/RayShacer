#include "hip/hip_runtime.h"
// rendering
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Rendering.hpp"
#include "../Utility/Color.hpp"
#include "../World/World.hpp"
#include "../Utility/vec2.hpp"


__device__ ShakColor render_pixel(const World& world, const vec2 coord, const player_info& player) {
	const auto [playerX,
		playerY,
		playerZ,
		playerYaw,
		playerPitch
	] = player;
	constexpr float FOV = 90.0f;

	// convert player angles to a looking vector
	// the 1/tan(fov/2) is the distance from the player to the screen
	float eyePos = (float)-1 / tan(toRadians(FOV / 2.0f));


	//vec3 rayDirection = (vec3(coord.x, coord.y, -eyePos)).normalize().rotateYP(playerYaw, playerPitch);
	vec3 rayDirection = (vec3(coord.x, coord.y, -eyePos))
		.normalize().rotateYP(playerYaw, playerPitch);

	ShakColor pixel;
	pixel.r = 0;
	pixel.g = 0;
	pixel.b = 0;
	pixel.a = 255;

	// color if it hits nothing
	vec3 normal = rayDirection * 0.5f + 0.5f;

	vec3 rayOrigin = vec3(
		playerX,
		playerY,
		playerZ
	);
	World::blockInfo block_info;
	constexpr int MAX_BOUNCES = 100;
	int i = 0;
	do {
		block_info = world.voxel_traversal_new(rayOrigin, rayDirection);
		float face{};
		vec3 trunc = block_info.where.fract();
		{
			// find face hit, its the one thats either 0.0001 or 0.999, aka the one thats closest to 0 or 1
			auto minval = std::min(std::min(trunc.x, trunc.y), trunc.z);
			auto maxval = std::max(std::max(trunc.x, trunc.y), trunc.z);
			// find which is closer, minval to 0, or maxval to 1
			// if minval is closer to 0, then the face is the minval
			// if maxval is closer to 1, then the face is the maxval
			auto normminval = minval - 0;
			auto normmaxval = (maxval - 1) * -1;
			face = normminval < normmaxval ? minval : maxval;
		}
		switch (block_info.type)
		{
		case blockType::Air:
			break;
		case blockType::Dirt:
		{
			if (face == trunc.x)
				pixel.r = trunc.z * 255;
			else if (face == trunc.y)
				pixel.g = trunc.x * 255;
			else
				pixel.b = trunc.y * 255;
			break;
		}
		case blockType::Stone:
			pixel.r = 100;
			pixel.g = 100;
			pixel.b = 100;
			break;

		case blockType::Mirror:
		{
			vec3 block_normal;

			// find the normal of the block

			if (face == trunc.x)
				block_normal = { 1,0,0 };
			else if (face == trunc.y)
				block_normal = { 0,1,0 };
			else
				block_normal = { 0,0,1 };

			if (block_info.where.x < rayOrigin.x ||
				block_info.where.y < rayOrigin.y ||
				block_info.where.z < rayOrigin.z)
				block_normal *= -1.0f;

			// reflect the ray
			rayDirection = rayDirection - (block_normal * 2.0f * rayDirection.dotThis(block_normal));
			normal = rayDirection * 0.5f + 0.5f;

			rayOrigin = block_info.where;
			break;

		}
		default:
			pixel.r = normal.x * 255;
			pixel.g = normal.y * 255;
			pixel.b = normal.z * 255;
			break;
		}

		++i;

		if (i >= MAX_BOUNCES) {
			pixel.r = 255;
			pixel.g = 255;
			pixel.b = 255;
		}
	} while (i < MAX_BOUNCES && block_info.type == Mirror);

	return pixel;
}

__global__ void render(World const * const world, const std::span<vec2> coords, std::span<ShakColor> colors, const player_info player) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < colors.size()) {
		vec2 coord = { (float)i / 256.0f, 0.5f };

		ShakColor pixel = render_pixel(*world, coord, player);

		// write to the screen
		colors[i] = pixel;
	}
}

void wrapper_thing(const World& world, const std::span<vec2> coords, std::span<ShakColor> colors, player_info &info) {

	vec2* d_coords;
	ShakColor* d_colors;
	World *d_world;

	// sending data to the gpu
	hipMalloc(&d_coords, coords.size_bytes());
	hipMalloc(&d_colors, colors.size_bytes());
	hipMalloc(&d_world, sizeof(World));

	hipMemcpy(d_coords, coords.data(), coords.size_bytes(), hipMemcpyHostToDevice);
	hipMemcpy(d_colors, colors.data(), colors.size_bytes(), hipMemcpyHostToDevice);
	hipMemcpy(d_world,  &world,		sizeof(World),		 hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	std::span<vec2> d_coords_s(d_coords, coords.size());
	std::span<ShakColor> d_colors_s(d_colors, colors.size());
	render <<<(colors.size() / 256 + 1), 256>>> (d_world, d_coords_s, d_colors_s, info);

	hipDeviceSynchronize();

	// get the data back
	hipMemcpy(colors.data(), d_colors, colors.size_bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(coords.data(), d_coords, coords.size_bytes(), hipMemcpyDeviceToHost);
	// we dont care about the world, it was not modified in the kernel

	hipDeviceSynchronize();

	hipFree(d_coords);
	hipFree(d_colors);
	hipFree(d_world);
}

